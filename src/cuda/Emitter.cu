#include "hip/hip_runtime.h"
#include "linux_helper.h"

#include "Emitter.cuh"
#include <Graphics.h>
#include <Camera.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <>

#include <iostream>

enum SHADER_ATTRIBUTES_IDX {
    POSITION = 0,
    SIZE = 1,
    TIME = 2,
    NUM_SHADER_ATTRIBUTES = 3
};

static std::string ShaderAttributes[NUM_SHADER_ATTRIBUTES] = {
        "positionIn",
        "sizeIn",
        "timeIn",
};

__global__ void initRand(hiprandState *_randstate) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // From CURAND library guide
    // Each thread gets same seed, a different sequence number
    // and no offset.

    hiprand_init(2345, tid, 0, &_randstate[tid]);
}

__global__ void init(Emitter::EmitterParams _p,
                     float *_time,
                     float *_pos,
                     float *_acc,
                     float *_vel,
                     float *_size) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < _p.numParticles_) {

        _time[tid] = 1.0f;

        _pos[3*tid+0] = _p.startPos_[0];
        _pos[3*tid+1] = _p.startPos_[1];
        _pos[3*tid+2] = _p.startPos_[2];

        _acc[3*tid+0] = _p.startAcc_[0];
        _acc[3*tid+1] = _p.startAcc_[1];
        _acc[3*tid+2] = _p.startAcc_[2];

        _vel[3*tid+0] = _p.startVel_[0];
        _vel[3*tid+1] = _p.startVel_[1];
        _vel[3*tid+2] = _p.startVel_[2];

        _size[tid] = _p.pointSize_;

        tid += blockDim.x * gridDim.x;
    }
}
                             
__global__ void newParticle(Emitter::EmitterParams _p,
                            float *_time,
                            float *_pos,
                            float *_acc,
                            float *_vel,
                            float *_size,
                            unsigned int _index,
                            hiprandState *_randstate) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int index;
    int limit;
    switch (_p.emitterType_) {
    case Emitter::EMITTER_STREAM:
        index = _index; // only add one new particle
        limit = _p.numParticles_; 
        break;
    case Emitter::EMITTER_BURST:
        index = tid; // add several particles
        if (_p.burstSize_ <= _p.numParticles_) limit = _p.burstSize_;
        else limit = _p.numParticles_;
        break;
    }

    while (index < limit) {

        // get three random floats for start velocity, one for time
        float vx_offset = 2.f * ( hiprand_normal(&_randstate[tid]) - 0.5f );
        float vy_offset = 2.f * ( hiprand_normal(&_randstate[tid]) - 0.5f );
        float vz_offset = 2.f * ( hiprand_normal(&_randstate[tid]) - 0.5f );
        float px_offset = 2.f * ( hiprand_normal(&_randstate[tid]) - 0.5f );
        float py_offset = 2.f * ( hiprand_normal(&_randstate[tid]) - 0.5f );
        float pz_offset = 2.f * ( hiprand_normal(&_randstate[tid]) - 0.5f );
        float t_offset = hiprand_normal(&_randstate[tid]);

        _time[index] = 1.0f + t_offset*0.01;

        _pos[3*index+0] = _p.startPos_[0] + px_offset * _p.posRandWeight_;
        _pos[3*index+1] = _p.startPos_[1] + py_offset * _p.posRandWeight_;
        _pos[3*index+2] = _p.startPos_[2] + pz_offset * _p.posRandWeight_;

        _acc[3*index+0] = _p.startAcc_[0];
        _acc[3*index+1] = _p.startAcc_[1];
        _acc[3*index+2] = _p.startAcc_[2];

        _vel[3*index+0] = _p.startVel_[0] + vx_offset * _p.velRandWeight_;
        _vel[3*index+1] = _p.startVel_[1] + vy_offset * _p.velRandWeight_;
        _vel[3*index+2] = _p.startVel_[2] + vz_offset * _p.velRandWeight_;

         _size[index] = _p.pointSize_;

        // only run once if stream (only add one at a time)
        if (_p.emitterType_ == Emitter::EMITTER_STREAM) break;

        index += blockDim.x * gridDim.x;

    }

}

__global__ void integrate(Emitter::EmitterParams _p,
                          float *_time,
                          float *_pos,
                          float *_acc,
                          float *_vel,
                          float *_size,
                          float _dt) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < _p.numParticles_) {

        if (_time[tid] > 0.0) {

            // subtract elapsed time
            _time[tid] -= (1.f/_p.lifeTime_)*_dt;

            _vel[3*tid+0] += _dt * _acc[3*tid+0];
            _vel[3*tid+1] += _dt * _acc[3*tid+1];
            _vel[3*tid+2] += _dt * _acc[3*tid+2];

            _pos[3*tid+0] += _dt * _vel[3*tid+0];
            _pos[3*tid+1] += _dt * _vel[3*tid+1];
            _pos[3*tid+2] += _dt * _vel[3*tid+2];

            _size[tid] *= _p.growthFactor_;

        }

        tid += blockDim.x * gridDim.x;
    }
}

Emitter::Emitter(unsigned int _numParticles, ShaderData*_sd) : shaderData_(_sd)
{

    // set standard values
    params_.numParticles_ = _numParticles;
    params_.burstSize_ = _numParticles;
    params_.emitterType_ = Emitter::EMITTER_STREAM;
    params_.growthFactor_ = 1.f;
    params_.lifeTime_ = 100.f;
    params_.mass_ = 1.f;
    params_.pointSize_ = 30.f;
    params_.posRandWeight_ = 0.f;
    params_.rate_ = 0.001f;
    params_.startAcc_[0] = 0.f;
    params_.startAcc_[1] = 0.f;
    params_.startAcc_[2] = 0.f;
    params_.startPos_[0] = 0.f;
    params_.startPos_[1] = 0.f;
    params_.startPos_[2] = 0.f;
    params_.startVel_[0] = 0.f;
    params_.startVel_[1] = 1.f;
    params_.startVel_[2] = 0.f;
    params_.velRandWeight_ = 0.f;
    params_.blendMode_ = Emitter::BLEND_FIRE;

    blocks_ = threads_ = 128;
    
    // allocate device memory
    hipMalloc((void**)&d_time_, sizeof(float)*_numParticles);
    hipMalloc((void**)&d_pos_, sizeof(float)*3*_numParticles);
    hipMalloc((void**)&d_acc_, sizeof(float)*3*_numParticles);
    hipMalloc((void**)&d_vel_, sizeof(float)*3*_numParticles);
    hipMalloc((void**)&d_size_, sizeof(float)*_numParticles);

    // for random states
    hipMalloc((void**)&d_randstate_, sizeof(hiprandState)*blocks_*threads_);

    // init
    init CUDA_KERNEL_DIM(blocks_,threads_)(params_,
                                           d_time_,
                                           d_pos_,
                                           d_acc_,
                                           d_vel_,
                                           d_size_);
    initRand CUDA_KERNEL_DIM(blocks_, threads_) (d_randstate_);

    // first particle goes in the first slot
    nextSlot_ = 0;

    // reset time
    nextEmission_ = params_.rate_;

    // generate VBOs
    std::string name("nat javla namn, todo");
    Graphics::instance().buffersNew(name, VAO_, vboPos_, vboSize_, vboTime_);
    Graphics::instance().geometryIs(
                                    vboPos_,
                                    vboSize_,
                                    vboTime_,
                                    _numParticles,
                                    VBO_DYNAMIC);

    const int id = shaderData_->shaderID();
    Graphics & g = Graphics::instance();

    int posLoc = g.shaderAttribLoc(id , ShaderAttributes[POSITION]);
    int sizeLoc = g.shaderAttribLoc(id , ShaderAttributes[SIZE]);
    int timeLoc = g.shaderAttribLoc(id , ShaderAttributes[TIME]);

    unsigned int sID = shaderData_->shaderID();

    g.bindGeometry(sID, VAO_, vboPos_, 3, 0, posLoc, 0);
    g.bindGeometry(sID, VAO_, vboSize_, 1, 0, sizeLoc, 0);
    g.bindGeometry(sID, VAO_, vboTime_, 1, 0, timeLoc, 0);

    cudaGLRegisterBufferObject(vboPos_);
    cudaGLRegisterBufferObject(vboSize_);
    cudaGLRegisterBufferObject(vboTime_);
}

void Emitter::display() const
{
    Matrix4 * modelView = shaderData_->stdMatrix4Data(MODELVIEW);
    Matrix4 * projection = shaderData_->stdMatrix4Data(PROJECTION);
    *modelView = Camera::instance().viewMtx();
    *projection = Camera::instance().projectionMtx();
    bool additive = params_.blendMode_ == BLEND_FIRE ? true : false;
    Graphics::instance().drawArrays(VAO_, params_.numParticles_,
            shaderData_, additive);
}

void Emitter::burst() {

    if (params_.emitterType_ != Emitter::EMITTER_BURST) return;

    cudaGLMapBufferObject((void**)&d_pos_, vboPos_);
    cudaGLMapBufferObject((void**)&d_size_, vboSize_);
    cudaGLMapBufferObject((void**)&d_time_, vboTime_);

    newParticle CUDA_KERNEL_DIM(blocks_,threads_)(params_,
                                                  d_time_,
                                                  d_pos_,
                                                  d_acc_,
                                                  d_vel_,
                                                  d_size_,
                                                  0,
                                                  d_randstate_);

    cudaGLUnmapBufferObject(vboPos_);
    cudaGLUnmapBufferObject(vboSize_);
    cudaGLUnmapBufferObject(vboTime_);
}

void Emitter::update(float _dt) {

    cudaGLMapBufferObject((void**)&d_pos_, vboPos_);
    cudaGLMapBufferObject((void**)&d_size_, vboSize_);
    cudaGLMapBufferObject((void**)&d_time_, vboTime_);

    // only care about new emissions if it's a stream
    if (params_.emitterType_ == Emitter::EMITTER_STREAM) {

        // count off elapsed time
        nextEmission_ -= _dt;

        // std::cout << "Next emission: " << nextEmission_ << std::endl;
         //std::cout << "Nect slot: " << nextSlot_ << std::endl;

        if (nextEmission_ < 0.0) {

            // calculate how many particles we should emit
            int numNewParticles = (int)(-nextEmission_/params_.rate_);

            // reset time for next emission
            nextEmission_ += numNewParticles*params_.rate_;
            nextEmission_ += params_.rate_;

            // emit new particles to make up for any overlap in elapsed time
            do {
                // emit a particle
                newParticle CUDA_KERNEL_DIM(1,1) (params_,
                                                  d_time_,
                                                  d_pos_,
                                                  d_acc_,
                                                  d_vel_,
                                                  d_size_,
                                                  nextSlot_,
                                                  d_randstate_);

                // jump forward one slot
                nextSlot_++;
                if (nextSlot_ == params_.numParticles_) nextSlot_ = 0;

                numNewParticles--;

            } while (numNewParticles > 0);
        
        } // if nextemission

    } // if stream
        
    // update all the particles
    integrate CUDA_KERNEL_DIM(blocks_,threads_) (params_,
                                                 d_time_,
                                                 d_pos_,
                                                 d_acc_,
                                                 d_vel_,
                                                 d_size_,
                                                 _dt);

    cudaGLUnmapBufferObject(vboPos_);
    cudaGLUnmapBufferObject(vboSize_);
    cudaGLUnmapBufferObject(vboTime_);

}

void Emitter::posIs(Vector3 _pos) {
    params_.startPos_[0] = _pos.x;
    params_.startPos_[1] = _pos.y;
    params_.startPos_[2] = _pos.z;
}

void Emitter::accIs(Vector3 _acc) {
    params_.startAcc_[0] = _acc.x;
    params_.startAcc_[1] = _acc.y;
    params_.startAcc_[2] = _acc.z;
}

void Emitter::velIs(Vector3 _vel) {
    params_.startVel_[0] = _vel.x;
    params_.startVel_[1] = _vel.y;
    params_.startVel_[2] = _vel.z;
}

void Emitter::rateIs(float _rate) {
    params_.rate_ = _rate;
}

void Emitter::massIs(float _mass) {
    params_.mass_ = _mass;
}

void Emitter::burstSizeIs(unsigned int _burstSize) {
    params_.burstSize_ = _burstSize;
}

void Emitter::lifeTimeIs(float _lifeTime) {
    params_.lifeTime_ = _lifeTime;
}

void Emitter::typeIs(Type _emitterType) {
    params_.emitterType_ = _emitterType;
}

void Emitter::pointSizeIs(float _size) {
    params_.pointSize_ = _size;
}
void Emitter::growthFactorIs(float _growthFactor) {
    params_.growthFactor_ = _growthFactor;
}

void Emitter::velRandWeightIs(float _velRandWeight) {
    params_.velRandWeight_ = _velRandWeight;
}

void Emitter::posRandWeightIs(float _posRandWeight) {
    params_.posRandWeight_ = _posRandWeight;
}

void Emitter::shaderDataIs(ShaderData * _shaderData) {
    shaderData_ = _shaderData;
}

void Emitter::blendModeIs(BlendMode _blendMode) {
    params_.blendMode_ = _blendMode;
}

void Emitter::copyPosToHostAndPrint() {

    float *h_pos;
    h_pos = new float[params_.numParticles_*3];
    hipMemcpy(h_pos, d_pos_, sizeof(float)*3*params_.numParticles_,
            hipMemcpyDeviceToHost);

    for (int i=0; i<params_.numParticles_; ++i) {
        std::cout << "(" << h_pos[3*i] << ", " << h_pos[3*i+1] << ", " << h_pos[3*i+2] << ")" << std::endl;
    }
    delete h_pos;

}






